#include <stdio.h>
#include <string>

#include "hip/hip_runtime.h"
#include "lib/helper_image.h"

unsigned char* goldenstandard(unsigned char* pixels) {
    unsigned char* output = NULL;

    return output;
}

int main(int argc, char** argv) {

    if(argc != 4) {
        std::cout << "Usage ./homework_5 <filter_size> <input_file> <output_file>" << std::endl;
        exit(1);
    }

    int filter_size = std::stoi(argv[1]);

    unsigned int w, h;
    unsigned char* pixels = NULL;

    if (sdkLoadPGM<unsigned char>(argv[2], &pixels, &w, &h) != true)
    {
        std::cout << "Unable to load PGM image file" << std::endl;
        return 1;
    }

    std::cout << "Successfully loaded PGM image file" << std::endl;

// First, create a 3x3 filter
	for (int i = 0; i < w; i++) {
        for (int j = 0; j < w; j++) {
            if (i > 0 && j > 0 && i < (w - 1) & j < (h - 1)) {
                std::vector<unsigned char> v;

                // Index of current pixel
                int k = i * w + j;

                // Top row of neighborhood
                v.push_back(pixels[k - (w - 1)]);
                v.push_back(pixels[k - (w)]);
                v.push_back(pixels[k - (w + 1))]);

                // Middle row of neighborhood
                v.push_back(pixels[k - 1]);
                v.push_back(pixels[k]);
                v.push_back(pixels[k + 1]);

                // Bottom row of neighborhood
                v.push_back(pixels[k + (w - 1)]);
                v.push_back(pixels[k + (w)]);
                v.push_back(pixels[k + (w + 1)]);

                std::sort(v.begin(), v.end());

                int size = v.size();
                pixels[k] = v[size / 2];
            }
        }
    }

    if (sdkSavePGM<unsigned char>(argv[3], pixels, w, h) != true)
    {
        std::cout << "Unable to save PGM image file" << std::endl;
        return 1;
    }

    std::cout << "Successfully saved PGM image file" << std::endl;

    return 0;
}
